#include "hip/hip_runtime.h"
#include "gpu_cloth.cuh"
#include "simulator.h"

__global__ 
void kernel_gravity_force(
    const glm::vec3 *X, const glm::vec3 *X_prev, const float *M, size_t size,
    const glm::vec3 gravity, float damping, float dt,
    glm::vec3 *F)
{
  const int pos = threadIdx.x + blockDim.x * blockIdx.x;

  if (pos < size) {
    F[pos] = gravity * M[pos] + damping * ((X[pos] - X_prev[pos]) / dt);
  }
}

__global__
void kernel_next_position(
    glm::vec3 *X, glm::vec3 *X_prev, glm::vec3 *X_prev_prev, 
    const glm::vec3 *F, const float *M, const float dt, size_t size)
{
  const int pos = threadIdx.x + blockDim.x * blockIdx.x;

  // TODO: stride and loop to optimise this, increasing thread computation load?
  if (pos < size) {
    glm::vec3 curr = X[pos];
    glm::vec3 prev = X_prev[pos];
    float m = M[pos];
    X[pos] = (m > 0.0000001f) ? (curr + curr - prev + dt * dt * F[pos] / m) : curr;
    X_prev_prev[pos] = prev;
    X_prev[pos] = curr;
  }
}

__global__
void kernel_spring_force(
    const CudaCloth::Spring *springs, const glm::vec3 *X, const glm::vec3 *X_prev,
    glm::vec3 *F,
    float dt, size_t size)
{
  const int pos = threadIdx.x + blockDim.x * blockIdx.x;
  
  if (pos < (size>>1)) {
    CudaCloth::Spring s = springs[pos];
    glm::vec3 p1 = X[s.p1];
    glm::vec3 p2 = X[s.p2];

    glm::vec3 spring_force = -s.ks * (glm::distance(p1, p2) - s.rest_length) * glm::normalize(p1 - p2);

    F[s.p1] += spring_force;
    F[s.p2] -= spring_force;

    s = springs[pos + (size>>1)];
    p1 = X[s.p1];
    p2 = X[s.p2];

    spring_force = -s.ks * (glm::distance(p1, p2) - s.rest_length) * glm::normalize(p1 - p2);

    F[s.p1] += spring_force;
    F[s.p2] -= spring_force;
  }
}


__global__
void kernel_satisfy_constraints(
    const CudaCloth::Spring *springs, const float *M, glm::vec3 *X,
    size_t size)
{
  const int pos = threadIdx.x + blockDim.x * blockIdx.x;

  if (pos < (size>>1)) {
    CudaCloth::Spring s = springs[pos];
    float m1 = M[s.p1];
    float m2 = M[s.p2];
    glm::vec3 p12 = X[s.p2] - X[s.p1];
    glm::vec3 correct_v = p12 * (1 - s.rest_length / glm::length(p12)) * 0.1f;
    if (m1 > 0.0000001f) X[s.p1] += correct_v;
    if (m2 > 0.0000001f) X[s.p2] -= correct_v;

    __syncthreads();
    s = springs[pos + (size>>1)];
    m1 = M[s.p1];
    m2 = M[s.p2];
    p12 = X[s.p2] - X[s.p1];
    correct_v = p12 * (1 - s.rest_length / glm::length(p12)) * 0.1f;
    if (m1 > 0.0000001f) X[s.p1] += correct_v;
    if (m2 > 0.0000001f) X[s.p2] -= correct_v;
  }
}

void CudaCloth::init()
{
  add_springs(KD_STRUCT, KS_STRUCT, KD_BEND, KS_BEND, KD_SHEAR, KS_SHEAR);
}

void CudaCloth::add_springs(
    float kd_struct, float ks_struct,
    float kd_bend, float ks_bend,
    float kd_shear, float ks_shear)
{
  // struct x
  for (uint32_t start = 0; start <= 1; ++start) {
    for (uint32_t y = 0; y < height; ++y) {
      for (uint32_t x = start; x < width-1; x += 2) {
        uint32_t p1 = x + width * y;
        uint32_t p2 = p1 + 1;
        springs_struct_x.push_back(Spring(p1, p2, glm::distance(X[p1], X[p2]), kd_struct, ks_struct));
      }
    }
  }

  // struct y
  for (uint32_t start = 0; start <= 1; ++start) {
    for (uint32_t x = 0; x < width; ++x) {
      for (uint32_t y = start; y < height-1; y += 2) {
        uint32_t p1 = x + width * y;
        uint32_t p2 = p1 + width;
        springs_struct_y.push_back(Spring(p1, p2, glm::distance(X[p1], X[p2]), kd_struct, ks_struct));
      }
    }
  }

  // shear
  for (uint32_t start = 0; start <= 1; ++start) {
    for (uint32_t y = start; y < height-1; y += 2) {
      for (uint32_t x = start; x < width-1; x += 2) {
        uint32_t p1 = x + width * y;
        uint32_t p2 = p1 + width + 1;
        springs_shear.push_back(Spring(p1, p2, glm::distance(X[p1], X[p2]), kd_shear, ks_shear));

        p1 += 1;
        p2 -= 1;
        springs_shear.push_back(Spring(p1, p2, glm::distance(X[p1], X[p2]), kd_shear, ks_shear));
      }
    }
  }

  // bend x
  for (uint32_t start = 0; start <= 2; start += 2) {
    for (uint32_t y = 0; y < height; ++y) {
      for (uint32_t x = start; x < width-2; x += 4) {
        uint32_t p1 = x + width * y;
        uint32_t p2 = p1 + 2;
        springs_bend_x.push_back(Spring(p1, p2, glm::distance(X[p1], X[p2]), kd_bend, ks_bend));

        p1 += 1;
        p2 += 1;
        springs_bend_x.push_back(Spring(p1, p2, glm::distance(X[p1], X[p2]), kd_bend, ks_bend));
      }
    }
  }

  // bend y
  for (uint32_t start = 0; start <= 2; start += 2) {
    for (uint32_t x = 0; x < width; ++x) {
      for (uint32_t y = start; y < height-2; y += 4) {
        uint32_t p1 = x + width * y;
        uint32_t p2 = p1 + width + width;
        springs_bend_y.push_back(Spring(p1, p2, glm::distance(X[p1], X[p2]), kd_bend, ks_bend));

        p1 += width;
        p2 += width;
        springs_bend_y.push_back(Spring(p1, p2, glm::distance(X[p1], X[p2]), kd_bend, ks_bend));
      }
    }
  }

  d_springs_shear = springs_shear;
  d_springs_struct_x = springs_struct_x;
  d_springs_struct_y = springs_struct_y;
  d_springs_bend_x = springs_bend_x;
  d_springs_bend_y = springs_bend_y;
}

void CudaCloth::timestep(float dt)
{
  satisfy_constraints();
  compute_forces(dt);
  compute_next_positions(dt);

  thrust::copy(d_X.begin(), d_X.end(), X.begin());
}

void CudaCloth::satisfy_constraints()
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  for (uint32_t i = 0; i < 10; ++i) {
    satisfy_spring_constraints(d_springs_struct_x);
    satisfy_spring_constraints(d_springs_struct_y);
    satisfy_spring_constraints(d_springs_shear);
    satisfy_spring_constraints(d_springs_bend_x);
    satisfy_spring_constraints(d_springs_bend_y);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  cout << "satisfy_constraints: " << milliseconds << "ms" << endl;
}

void CudaCloth::satisfy_spring_constraints(const thrust::device_vector<Spring>& springs)
{
  kernel_satisfy_constraints<<<(springs.size()/2 + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>
    (thrust::raw_pointer_cast(springs.data()),
     thrust::raw_pointer_cast(d_M.data()),
     thrust::raw_pointer_cast(d_X.data()),
     springs.size());
}

void CudaCloth::compute_forces(float dt)
{

  kernel_gravity_force<<<(d_X.size() + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>
    (thrust::raw_pointer_cast(d_X.data()),
     thrust::raw_pointer_cast(d_X_prev.data()),
     thrust::raw_pointer_cast(d_M.data()), d_X.size(),
     GRAVITY, DAMPING, dt,
     thrust::raw_pointer_cast(d_F.data()));

  compute_spring_forces(d_springs_struct_x, dt);
  compute_spring_forces(d_springs_struct_y, dt);
  compute_spring_forces(d_springs_bend_x, dt);
  compute_spring_forces(d_springs_bend_y, dt);
  compute_spring_forces(d_springs_shear, dt);
}

void CudaCloth::compute_next_positions(float dt)
{
  kernel_next_position<<<(X.size() + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>
    (thrust::raw_pointer_cast(d_X.data()),
     thrust::raw_pointer_cast(d_X_prev.data()),
     thrust::raw_pointer_cast(d_X_prev_prev.data()),
     thrust::raw_pointer_cast(d_F.data()),
     thrust::raw_pointer_cast(d_M.data()),
     dt, X.size());
}

void CudaCloth::compute_spring_forces(const thrust::device_vector<Spring>& springs, float dt)
{
  kernel_spring_force<<<(springs.size()/2 + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>
    (thrust::raw_pointer_cast(springs.data()),
     thrust::raw_pointer_cast(d_X.data()),
     thrust::raw_pointer_cast(d_X_prev.data()),
     thrust::raw_pointer_cast(d_F.data()),
     dt, springs.size());
}

void CudaCloth::compute_wind_forces()
{
}

void CudaCloth::compute_self_collisions()
{
}

void CudaCloth::compute_object_collisions()
{
}


Simulator::Simulator(bool gpu)
{
  if (gpu) {
    cloth = new CudaCloth(50, 50, 0.5, 0.5);
  } else {
    cloth = new Cloth(10, 10, 0.5, 0.5);
  }
  cloth->init();
}

void Simulator::compute_timestep(float dt)
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  cloth->timestep(dt);
  
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  cout << "overall: " << milliseconds << "ms" << endl;
}

void Simulator::draw()
{
  cloth->draw();
}

